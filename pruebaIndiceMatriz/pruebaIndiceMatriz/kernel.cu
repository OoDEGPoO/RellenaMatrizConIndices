
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t RellenaMatriz(int *x, int *y, int *m, unsigned int sizeX, unsigned int sizeY);

void imprimeVector(int *v, int n) {
	printf("{");
	for (int i = 1; i <= n; i++) {
		printf("%d", *v);
		if (i != n) printf(", ");
		v++;
	}
	printf("}");
}

void imprimeMatriz(int *v, int m, int n) {
	int i, j;
	printf("\n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			printf("%d\t", v[i*n+j]);
		}
		printf("\n");
	}
}

__global__ void rmKernel(int *x, int *y, int *m) {
	int idx = threadIdx.x;
	int idy = blockIdx.x;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	x[idx] = idx;
	y[idy] = idy;
	m[id] = idy*10 + idx;
}

int main()
{
    const int sizeX = 5;
	const int sizeY = 6;
    int x[sizeX] = { 0, 0, 0, 0, 0 };
    int y[sizeY] = { 0, 0, 0, 0, 0, 0 };

	int m[sizeY*sizeX] =	{ 0, 0, 0, 0, 0 
							, 0, 0, 0, 0, 0
							, 0, 0, 0, 0, 0
							, 0, 0, 0, 0, 0
							, 0, 0, 0, 0, 0
							, 0, 0, 0, 0, 0 };

    // Add vectors in parallel.
	hipError_t cudaStatus = RellenaMatriz(x, y, m, sizeX, sizeY);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Fallo en RellenaMatriz");
        return 1;
    }

	imprimeVector(x, sizeX);
	imprimeVector(y, sizeY);
	imprimeMatriz(m, sizeY, sizeX);


    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t RellenaMatriz(int *x, int *y, int *m, unsigned int sizeX, unsigned int sizeY)
{
    int *dev_x = 0;
    int *dev_y = 0;
	int *dev_m = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_x, sizeX * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_y, sizeY * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_m, sizeY * sizeX * sizeof(int *));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	/*
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = cudaMemcpy(dev_x, x, sizeX * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

    cudaStatus = cudaMemcpy(dev_y, y, sizeY * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }*/

    // Launch a kernel on the GPU with one thread for each element.
	rmKernel <<<sizeY, sizeX>>>(dev_x, dev_y, dev_m);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(x, dev_x, sizeX * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(y, dev_y, sizeY * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(m, dev_m, sizeY * sizeX * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
    hipFree(dev_x);
    hipFree(dev_y);
	hipFree(dev_m);
    
    return cudaStatus;
}
